#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//Thrust
#include <odl_cpp_utils/cuda/disableThrustWarnings.h>
#include <thrust/device_vector.h>
#include <odl_cpp_utils/cuda/enableThrustWarnings.h>

namespace gpumci {
namespace cuda {
//Initialize the hiprandState vector
__global__ void
curand_init_Kernel(unsigned seed, unsigned offset, unsigned numThreads, unsigned thread_offset, hiprandState_t* state) {
    unsigned idx = thread_offset + (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= numThreads)
        return;

    hiprand_init(seed, idx, offset, &state[idx]);
}

//A non-safe version of the init code. This one is significantly (several orders of magnitude) faster, but does not have
//as good statistical guarantees
// This general template works for hiprandStateMRG32k3a_t, hiprandStatePhilox4_32_10_t and hiprandStateXORWOW_t
__global__ void
curand_init_fast_Kernel(unsigned seed, unsigned offset, unsigned numThreads, unsigned thread_offset, hiprandState_t* state) {
    unsigned idx = thread_offset + (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= numThreads)
        return;

    hiprand_init(seed * numThreads + idx, 0, offset, &state[idx]);
}
}
}
