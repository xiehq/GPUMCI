#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cassert>

#include <float.h>
#include <cstdio>
#include <odl_cpp_utils/cuda/cutil_math.h>
#include <odl_cpp_utils/cuda/cuda_utils.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <>

// thrust
#include <thrust/device_vector.h>
#include <odl_cpp_utils/cuda/texture.h>

namespace gpumci {
namespace cuda {

__global__ void cosWeightingKernel(const float3 sourcePosition,
                                   const float3 detectorOrigin,
                                   const float3 pixelDirectionU,
                                   const float3 pixelDirectionV,
                                   const int2 detectorSize,
                                   float* source,
                                   float* target) {
    const uint2 id{blockIdx.x * blockDim.x + threadIdx.x,
                   blockIdx.y * blockDim.y + threadIdx.y};

    if (id.x >= detectorSize.x ||
        id.y >= detectorSize.y)
        return;

    //Add 0.5 to center the pixels
    const float3 pixelPosition = detectorOrigin + pixelDirectionU * (uint2float(id.x) + 0.5f) + pixelDirectionV * (uint2float(id.y) + 0.5f);
    const float3 direction = normalize(pixelPosition - sourcePosition);

    float dir_dot = fabsf(dot(direction, normalize(cross(pixelDirectionU, pixelDirectionV))));

    target[id.x + id.y * detectorSize.x] = source[id.x + id.y * detectorSize.x] / dir_dot;
}
}

void apply_cosweighting(const float3 sourcePosition,
                        const float3 detectorOrigin,
                        const float3 pixelDirectionU,
                        const float3 pixelDirectionV,
                        const int2 detectorSize,
                        float* source,
                        float* target) {
    dim3 dimBlock(16, 16);
    dim3 dimGrid(static_cast<unsigned int>(1 + (detectorSize.x / dimBlock.x)),
                 static_cast<unsigned int>(1 + (detectorSize.y / dimBlock.y)));

    cuda::cosWeightingKernel<<<dimGrid, dimBlock>>>(sourcePosition,
                                                    detectorOrigin,
                                                    pixelDirectionU,
                                                    pixelDirectionV,
                                                    detectorSize,
                                                    source,
                                                    target);
    CUDA_CHECK_ERRORS;
    gpuErrchk(hipDeviceSynchronize());
}

} //gpumci
