#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>
#include <iostream>
#include <algorithm>

#include <odl_cpp_utils/cuda/disableThrustWarnings.h>
#include <thrust/device_vector.h>
#include <odl_cpp_utils/cuda/enableThrustWarnings.h>

#include <GPUMCI/implementations/PrecomputedMC.h>

#include <GPUMCI/physics/MaterialEntry.h>
#include <GPUMCI/physics/CudaSettings.h>
#include <GPUMCI/implementations/MaterialUtils.cuh>
#include <GPUMCI/implementations/WoodcockUtils.cuh>

//CudaMonteCarlo parts
#include <GPUMCI/detector/DetectorCBCTScatter.cuh>
#include <GPUMCI/photongenerator/PhotonGeneratorUniform.cuh>
#include <GPUMCI/rng/CurandRng.cuh>
#include <GPUMCI/interactions/PhotonPhoto.cuh>
#include <GPUMCI/interactions/InteractionHandlerPhoton.cuh>
#include <GPUMCI/interactions/ComptonPrecomputed.cuh>
#include <GPUMCI/interactions/RayleighPrecomputed.cuh>
#include <GPUMCI/interactions/WoodcockStep.cuh>

#include <GPUMCI/physics/CudaMonteCarlo.cuh>

#include <odl_cpp_utils/utils/cast.h>
#include <odl_cpp_utils/cuda/texture.h>

namespace gpumci {
namespace cuda {
namespace {
unsigned nThreads(int2 detectorSize) {
    return detectorSize.x * detectorSize.y;
}
}

//Struct that holds all data needed for the cuda MC simulation
struct PrecomputedMCCuData {
    PrecomputedMCCuData(const int3 volumeSize,
                        const int2 detectorSize,
                        const MaterialData& attenuationData_,
                        const InteractionTables& rayleighTables,
                        const InteractionTables& comptonTables)
        : attenuationData(attenuationData_),
          densityVolume(std::make_shared<BoundTexture3D<float>>(volumeSize,
                                                                hipAddressModeClamp,
                                                                hipFilterModeLinear,
                                                                hipReadModeElementType)),
          materialTypeVolume(std::make_shared<BoundTexture3D<uint8_t>>(volumeSize,
                                                                       hipAddressModeClamp,
                                                                       hipFilterModePoint,
                                                                       hipReadModeElementType)),
          rng(nThreads(detectorSize)),
          rayleigh(rayleighTables),
          compton(comptonTables) { //detectorSize.x * detectorSize.y) {

        int n_energy = narrow_cast<int>(attenuationData.n_energies);
        int n_materials = narrow_cast<int>(attenuationData.n_materials);

        //Interaction
        thrust::device_vector<float4> data = util::make_material_device(attenuationData);
        texMaterial = std::make_shared<BoundTexture2D<float4>>(int2{n_energy, n_materials},
                                                               hipAddressModeClamp,
                                                               hipFilterModeLinear,
                                                               hipReadModeElementType);
        texMaterial->setData(thrust::raw_pointer_cast(&data[0]));
    }

    //Nocopy
    PrecomputedMCCuData(const PrecomputedMCCuData&) = delete;
    PrecomputedMCCuData& operator=(const PrecomputedMCCuData&) = delete;

    const MaterialData attenuationData;
    std::shared_ptr<BoundTexture3D<float>> densityVolume;
    std::shared_ptr<BoundTexture3D<uint8_t>> materialTypeVolume;
    std::shared_ptr<BoundTexture2D<float4>> texMaterial;
    std::shared_ptr<WoodcockStep> woodcockStep;
    const RayleighPrecomputed rayleigh;
    const ComptonPrecomputed compton;
    curandRng rng;
};
}

PrecomputedMC::PrecomputedMC(const Eigen::Vector3i& volumeSize,
                             const Eigen::Vector3d& volumeOrigin,
                             const Eigen::Vector3d& voxelSize,
                             const Eigen::Vector2i& detectorSize,
                             int n_runs,
                             const MaterialData& attenuationData,
                             const InteractionTables& rayleighTables,
                             const InteractionTables& comptonTables)
    : _param{volumeSize, volumeOrigin, voxelSize, attenuationData.energyStep},
      _detectorSize(detectorSize),
      _nRuns(n_runs) {
    // Initialize the cuda side
    _cudaData = std::make_shared<cuda::PrecomputedMCCuData>(make_int3(volumeSize),
                                                            make_int2(detectorSize),
                                                            attenuationData,
                                                            rayleighTables,
                                                            comptonTables);
}

void PrecomputedMC::setData(const float* densityDevice,
                            const uint8_t* materialTypeDevice) {
    //Set the density and materials
    _cudaData->densityVolume->setData(densityDevice);
    _cudaData->materialTypeVolume->setData(materialTypeDevice);

    //Since the densities have updated, we need to update the woodcock table
    int n_energy = narrow_cast<int>(_cudaData->attenuationData.n_energies);
    _cudaData->woodcockStep = std::make_shared<cuda::WoodcockStep>(densityDevice,
                                                                   materialTypeDevice,
                                                                   _param.volumeSize,
                                                                   n_energy,
                                                                   _param.invEnergyStep,
                                                                   _cudaData->attenuationData);
}

void PrecomputedMC::project(const Eigen::Vector3d& sourcePosition,
                            const Eigen::Vector3d& detectorOrigin,
                            const Eigen::Vector3d& pixelDirectionU,
                            const Eigen::Vector3d& pixelDirectionV,
                            float* primary,
                            float* scatter) const {
    // Setup kernel configuration
    unsigned numberOfThreads = cuda::nThreads(make_int2(_detectorSize));
    float2 inversePixelSize = make_float2(1.0f / (float)pixelDirectionU.norm(),
                                          1.0f / (float)pixelDirectionV.norm());

    // Create a detector
    cuda::DetectorCBCTScatter detector{make_float3(detectorOrigin),
                                       make_float3(pixelDirectionU),
                                       make_float3(pixelDirectionV),
                                       inversePixelSize,
                                       make_int2(_detectorSize),
                                       narrow_cast<unsigned>(_detectorSize[0]),
                                       primary,
                                       scatter};

    //Use a analytic photon generator
    cuda::PhotonGeneratorUniform photonGenerator{make_int2(_detectorSize),
                                                 make_float3(detectorOrigin),
                                                 make_float3(pixelDirectionU),
                                                 make_float3(pixelDirectionV),
                                                 make_float3(sourcePosition),
                                                 _nRuns};

    //Simple interaction handler
    auto interaction = cuda::makePhotonInteractionHandler(_cudaData->compton.deviceSide(),
                                                          _cudaData->rayleigh.deviceSide(),
                                                          cuda::PhotonPhoto{},
                                                          _cudaData->texMaterial->tex());

    cuda::RunMC(_cudaData->densityVolume->tex(),
                _cudaData->materialTypeVolume->tex(),
                _param,
                numberOfThreads,
                interaction,
                photonGenerator,
                detector,
                _cudaData->woodcockStep->deviceSide(),
                _cudaData->rng.deviceSide());
}
}
