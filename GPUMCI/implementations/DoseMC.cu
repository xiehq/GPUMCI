#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>
#include <iostream>
#include <algorithm>

#include <odl_cpp_utils/cuda/disableThrustWarnings.h>
#include <thrust/device_vector.h>
#include <odl_cpp_utils/cuda/enableThrustWarnings.h>

#include <GPUMCI/implementations/DoseMC.h>

#include <GPUMCI/physics/MaterialEntry.h>
#include <GPUMCI/physics/CudaSettings.h>
#include <GPUMCI/implementations/MaterialUtils.cuh>
#include <GPUMCI/implementations/WoodcockUtils.cuh>

//CudaMonteCarlo parts
#include <GPUMCI/detector/DetectorCBCTScatter.cuh>
#include <GPUMCI/detector/DoseDetector.cuh>
#include <GPUMCI/photongenerator/PhotonGeneratorUniform.cuh>
#include <GPUMCI/rng/CurandRng.cuh>
#include <GPUMCI/interactions/PhotonPhoto.cuh>
#include <GPUMCI/interactions/InteractionHandlerPhoton.cuh>
#include <GPUMCI/interactions/ComptonPrecomputed.cuh>
#include <GPUMCI/interactions/RayleighPrecomputed.cuh>
#include <GPUMCI/interactions/WoodcockStep.cuh>

#include <GPUMCI/physics/CudaMonteCarlo.cuh>

#include <odl_cpp_utils/utils/cast.h>
#include <odl_cpp_utils/cuda/texture.h>

namespace gpumci {
namespace cuda {
namespace {
unsigned nThreads(int2 detectorSize) {
    return detectorSize.x * detectorSize.y;
}
}

//Struct that holds all data needed for the cuda MC simulation
struct DoseMCCuData {
    DoseMCCuData(const int3 volumeSize,
                 const int2 detectorSize_,
                 const MaterialData& attenuationData_,
                 const InteractionTables& rayleighTables,
                 const InteractionTables& comptonTables)
        : detectorSize(detectorSize_),
          attenuationData(attenuationData_),
          densityVolume(std::make_shared<BoundTexture3D<float>>(volumeSize,
                                                                hipAddressModeClamp,
                                                                hipFilterModeLinear,
                                                                hipReadModeElementType)),
          materialTypeVolume(std::make_shared<BoundTexture3D<uint8_t>>(volumeSize,
                                                                       hipAddressModeClamp,
                                                                       hipFilterModePoint,
                                                                       hipReadModeElementType)),
          rng(nThreads(detectorSize)),
          rayleigh(rayleighTables),
          compton(comptonTables) { //detectorSize.x * detectorSize.y) {

        int n_energy = narrow_cast<int>(attenuationData.n_energies);
        int n_materials = narrow_cast<int>(attenuationData.n_materials);

        //Interaction
        thrust::device_vector<float4> data = util::make_material_device(attenuationData);
        texMaterial = std::make_shared<BoundTexture2D<float4>>(int2{n_energy, n_materials},
                                                               hipAddressModeClamp,
                                                               hipFilterModeLinear,
                                                               hipReadModeElementType);
        texMaterial->setData(thrust::raw_pointer_cast(&data[0]));
    }

    //Nocopy
    DoseMCCuData(const DoseMCCuData&) = delete;
    DoseMCCuData& operator=(const DoseMCCuData&) = delete;

    const int2 detectorSize;
    const MaterialData attenuationData;
    std::shared_ptr<BoundTexture3D<float>> densityVolume;
    std::shared_ptr<BoundTexture3D<uint8_t>> materialTypeVolume;
    std::shared_ptr<BoundTexture2D<float4>> texMaterial;
    std::shared_ptr<WoodcockStep> woodcockStep;
    const RayleighPrecomputed rayleigh;
    const ComptonPrecomputed compton;
    curandRng rng;
};
}
DoseMC::DoseMC(const Eigen::Vector3i& volumeSize,
               const Eigen::Vector3d& volumeOrigin,
               const Eigen::Vector3d& voxelSize,
               const Eigen::Vector2i& detectorSize,
               int n_runs,
               const MaterialData& attenuationData,
               const InteractionTables& rayleighTables,
               const InteractionTables& comptonTables)
    : _param{volumeSize, volumeOrigin, voxelSize, attenuationData.energyStep},
      _nRuns(n_runs) {
    // Initialize the cuda side
    _cudaData = std::make_shared<cuda::DoseMCCuData>(make_int3(volumeSize),
                                                     make_int2(detectorSize),
                                                     attenuationData,
                                                     rayleighTables,
                                                     comptonTables);
}

void DoseMC::setData(const float* densityDevice,
                     const uint8_t* materialTypeDevice) {
    //Set the density and materials
    _cudaData->densityVolume->setData(densityDevice);
    _cudaData->materialTypeVolume->setData(materialTypeDevice);

    //Since the densities have updated, we need to update the woodcock table
    int n_energy = narrow_cast<int>(_cudaData->attenuationData.n_energies);
    _cudaData->woodcockStep = std::make_shared<cuda::WoodcockStep>(densityDevice,
                                                                   materialTypeDevice,
                                                                   _param.volumeSize,
                                                                   n_energy,
                                                                   _param.invEnergyStep,
                                                                   _cudaData->attenuationData);
}

void DoseMC::project(const Eigen::Vector3d& sourcePosition,
                     const Eigen::Vector3d& detectorOrigin,
                     const Eigen::Vector3d& pixelDirectionU,
                     const Eigen::Vector3d& pixelDirectionV,
                     float* primary,
                     float* scatter,
                     float* dose_volume) const {
    // Setup kernel configuration
    unsigned numberOfThreads = cuda::nThreads(_cudaData->detectorSize);
    float2 inversePixelSize = make_float2(1.0f / (float)pixelDirectionU.norm(),
                                          1.0f / (float)pixelDirectionV.norm());

    // Create a detector
    cuda::DetectorCBCTScatter detector{make_float3(detectorOrigin),
                                       make_float3(pixelDirectionU),
                                       make_float3(pixelDirectionV),
                                       inversePixelSize,
                                       _cudaData->detectorSize,
                                       narrow_cast<unsigned>(_cudaData->detectorSize.x),
                                       primary,
                                       scatter};

    //Use a analytic photon generator
    cuda::PhotonGeneratorUniform photonGenerator{_cudaData->detectorSize,
                                                 make_float3(detectorOrigin),
                                                 make_float3(pixelDirectionU),
                                                 make_float3(pixelDirectionV),
                                                 make_float3(sourcePosition),
                                                 _nRuns};

    //Simple interaction handler
    auto interaction = cuda::makePhotonInteractionHandler(_cudaData->compton.deviceSide(),
                                                          _cudaData->rayleigh.deviceSide(),
                                                          cuda::PhotonPhoto{},
                                                          _cudaData->texMaterial->tex());

    cuda::DoseDetector<decltype(interaction)> dose_interaction{_param.volumeMin,
                                                               _param.inverseVoxelSize,
                                                               _param.volumeSize,
                                                               dose_volume,
                                                               interaction};

    RunMC(_cudaData->densityVolume->tex(),
          _cudaData->materialTypeVolume->tex(),
          _param,
          numberOfThreads,
          dose_interaction,
          photonGenerator,
          detector,
          _cudaData->woodcockStep->deviceSide(),
          _cudaData->rng.deviceSide());
}
}
